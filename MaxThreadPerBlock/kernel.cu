﻿#include <iostream>
#include <hip/hip_runtime.h>



int main()
{
    // 獲取當前 CUDA 設備的 ID
    int deviceId;
    hipGetDevice(&deviceId);

    // 顯示當前使用的設備 ID
    std::cout << "Device: " << deviceId << std::endl;

    // 獲取硬體的靜態限制每個線程塊可容納的最大線程數
    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, deviceId);
    std::cout << "Max Threads Per Block :" << maxThreadsPerBlock << std::endl;

    // 獲取每個線程塊的最大維度
    int maxThreadsDimX, maxThreadsDimY, maxThreadsDimZ;
    hipDeviceGetAttribute(&maxThreadsDimX, hipDeviceAttributeMaxBlockDimX, deviceId);
    hipDeviceGetAttribute(&maxThreadsDimY, hipDeviceAttributeMaxBlockDimY, deviceId);
    hipDeviceGetAttribute(&maxThreadsDimZ, hipDeviceAttributeMaxBlockDimZ, deviceId);

    std::cout << "Max Threads Dim X: " << maxThreadsDimX << std::endl;
    std::cout << "Max Threads Dim Y: " << maxThreadsDimY << std::endl;
    std::cout << "Max Threads Dim Z: " << maxThreadsDimZ << std::endl;

    // 獲取每個網格的最大維度
    int maxGridDimX, maxGridDimY, maxGridDimZ;
    hipDeviceGetAttribute(&maxGridDimX, hipDeviceAttributeMaxGridDimX, deviceId);
    hipDeviceGetAttribute(&maxGridDimY, hipDeviceAttributeMaxGridDimY, deviceId);
    hipDeviceGetAttribute(&maxGridDimZ, hipDeviceAttributeMaxGridDimZ, deviceId);

    std::cout << "Max Grid Dim X: " << maxGridDimX << std::endl;
    std::cout << "Max Grid Dim Y: " << maxGridDimY << std::endl;
    std::cout << "Max Grid Dim Z: " << maxGridDimZ << std::endl;

    // 獲取每個線程束的大小
    int warpSize;
    hipDeviceGetAttribute(&warpSize, hipDeviceAttributeWarpSize, deviceId);

    std::cout << "Warp Size: " << warpSize << std::endl;

    return 0;
}

