﻿#include <iostream>
#include <hip/hip_runtime.h>

using namespace std;

int main()
{
    // 獲取當前 CUDA 設備的 ID
    int deviceId;
    hipGetDevice(&deviceId);

    // 顯示當前使用的設備 ID
    cout << "Device: " << deviceId << endl;

    // 獲取硬體的靜態限制每個線程塊可容納的最大線程數
    int maxThreadsPerBlock;
    hipDeviceGetAttribute(&maxThreadsPerBlock, hipDeviceAttributeMaxThreadsPerBlock, deviceId);
    cout << "Max Threads Per Block :" << maxThreadsPerBlock << endl;

    // 獲取每個線程塊的最大維度
    int maxThreadsDimX, maxThreadsDimY, maxThreadsDimZ;
    hipDeviceGetAttribute(&maxThreadsDimX, hipDeviceAttributeMaxBlockDimX, deviceId);
    hipDeviceGetAttribute(&maxThreadsDimY, hipDeviceAttributeMaxBlockDimY, deviceId);
    hipDeviceGetAttribute(&maxThreadsDimZ, hipDeviceAttributeMaxBlockDimZ, deviceId);

    cout << "Max Threads Dim X: " << maxThreadsDimX << endl;
    cout << "Max Threads Dim Y: " << maxThreadsDimY << endl;
    cout << "Max Threads Dim Z: " << maxThreadsDimZ << endl;

    // 獲取每個網格的最大維度
    int maxGridDimX, maxGridDimY, maxGridDimZ;
    hipDeviceGetAttribute(&maxGridDimX, hipDeviceAttributeMaxGridDimX, deviceId);
    hipDeviceGetAttribute(&maxGridDimY, hipDeviceAttributeMaxGridDimY, deviceId);
    hipDeviceGetAttribute(&maxGridDimZ, hipDeviceAttributeMaxGridDimZ, deviceId);

    cout << "Max Grid Dim X: " << maxGridDimX << endl;
    cout << "Max Grid Dim Y: " << maxGridDimY << endl;
    cout << "Max Grid Dim Z: " << maxGridDimZ << endl;

    // 獲取每個線程束的大小
    int warpSize;
    hipDeviceGetAttribute(&warpSize, hipDeviceAttributeWarpSize, deviceId);

    cout << "Warp Size: " << warpSize << endl;

    return 0;
}

